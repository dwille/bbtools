#include "hip/hip_runtime.h"
#include "cuda_sort.h"

// Fill the particle bins arrays -- partBin and partInd
__global__ void bin_fill(int *partInd, int *partBin, int nparts,
  part_struct *parts, dom_struct *binDom, BC bc) 
{
  int pp = threadIdx.x + blockIdx.x*blockDim.x;;

  int c;
  int ibin, jbin, kbin;

  // Find the correct bin index for each part and store it
  if (pp < nparts) {
    ibin = floor((parts[pp].x - binDom->xs)/binDom->dx);
    jbin = floor((parts[pp].y - binDom->ys)/binDom->dy);
    kbin = floor((parts[pp].z - binDom->zs)/binDom->dz);
    c = ibin + jbin*binDom->Gcc.s1 + kbin*binDom->Gcc.s2;

    partInd[pp] = pp;         // index of particle
    partBin[pp] = c;          // bin index
    parts[pp].bin = c;        // bin index (stored in particle)
  }
}

__global__ void init(int *array, int length, int val)
{
  int pp = threadIdx.x + blockIdx.x*blockDim.x;;
  if (pp < length)  {
    array[pp] =  val;
  }
}

__global__ void bin_start(int *binStart, int *binEnd, int *partBin, int nparts)
{
  // This kernel function was adapted from NVIDIA CUDA 5.5 Examples
  // This software contains source code provided by NVIDIA Corporation
  extern __shared__ int sharedBin[];    //blockSize + 1
  int index = threadIdx.x + blockIdx.x*blockDim.x;
  int bin;

  // for a given bin index, the previous bins's index is stored in sharedBin
  if (index < nparts) {
    bin = partBin[index]; 

    // Load bin data into shared memory so that we can look
    // at neighboring particle's hash value without loading
    // two bin values per thread
    sharedBin[threadIdx.x + 1] = bin;

    if (index > 0 && threadIdx.x == 0) {
      // first thread in block must load neighbor particle bin
      sharedBin[0] = partBin[index - 1];
    }
  }
  __syncthreads();

  if (index < nparts) {
    // If this particle has a different cell index to the previous
    // particle then it must be the first particle in the cell,
    // so store the index of this particle in the cell.
    // As it isn't the first particle, it must also be the cell end of
    // the previous particle's cell
    bin = partBin[index]; 

    if (index == 0 || bin != sharedBin[threadIdx.x]) {
    binStart[bin] = index;

        if (index > 0)
            binEnd[sharedBin[threadIdx.x]] = index;
    }

    if (index == nparts - 1)
    {
        binEnd[bin] = index + 1;
    }
  }
}

__global__ void find_nodes(part_struct *parts, int nparts, dom_struct *dom, 
  BC bc, int *binStart, int *binEnd, int *partBin, int *partInd, 
  dom_struct *binDom, int *neighborList, int *neighborCount, int nMax)
{
  int index = threadIdx.x + blockIdx.x*blockDim.x;

  if (index < nparts) {
    int i = partInd[index];
    int bin = partBin[index];

    int kbin = floorf(bin/binDom->Gcc.s2);
    int jbin = floorf((bin - kbin*binDom->Gcc.s2)/binDom->Gcc.s1);
    int ibin = bin - kbin*binDom->Gcc.s2 - jbin*binDom->Gcc.s1;

    int l, m, n;                          // adjacent bin iterators
    int target, j;                        // target indices
    int adjBin, adjStart, adjEnd;         // adjacent bin stuff
    int iStride, kStride, jStride;        // how to get to Sesame Street

    int cc = 0;                           // counter for inner neighborlist

    // predefine face locations 
    // -1, -2 due to local vs global indexing and defiinition of dom_struct
    int fW = binDom->Gcc.is - 1;
    int fE = binDom->Gcc.ie - 2;
    int fS = binDom->Gcc.js - 1;
    int fN = binDom->Gcc.je - 2;
    int fB = binDom->Gcc.ks - 1;
    int fT = binDom->Gcc.ke - 2;

    // size checks
    int xnBin = (binDom->xn > 2);
    int ynBin = (binDom->yn > 2);
    int znBin = (binDom->zn > 2);

    // loop over adjacent bins and take care of periodic conditions 
    for (n = -1; n <= 1; n++) {
      // if on a face and not periodic, continue
      // if on a face and periodic but only 2 bins, continue
      if ((n == -1 && kbin == fB && bc.pB != PERIODIC) || 
          (n ==  1 && kbin == fT && bc.pT != PERIODIC) ||
          (n == -1 && kbin == fB && bc.pB == PERIODIC && znBin == 0) ||
          (n ==  1 && kbin == fT && bc.pT == PERIODIC && znBin == 0)) {
        continue;
      // if on a face and periodic, flip to other side
      } else if (n == -1 && kbin == fB && bc.pB == PERIODIC) {
        kStride = fT*binDom->Gcc.s2;
      } else if (n ==  1 && kbin == fT && bc.pT == PERIODIC) {
        kStride = fB*binDom->Gcc.s2;
      // else, we are in the middle, do nothing special
      } else {
        kStride = (kbin + n)*binDom->Gcc.s2;
      }

      for (m = -1; m <= 1; m++) {
        if ((m == -1 && jbin == fS && bc.pS != PERIODIC) ||
            (m ==  1 && jbin == fN && bc.pN != PERIODIC) ||
            (m == -1 && jbin == fS && bc.pS == PERIODIC && ynBin == 0) ||
            (m ==  1 && jbin == fN && bc.pN == PERIODIC && ynBin == 0)) {
          continue;
        } else if (m == -1 && jbin == fS && bc.pS == PERIODIC) {
          jStride = fN*binDom->Gcc.s1;  
        } else if (m ==  1 && jbin == fN && bc.pN == PERIODIC) {
          jStride = fS*binDom->Gcc.s1;
        } else {
          jStride = (jbin + m)*binDom->Gcc.s1;
        }

        for (l = -1; l <= 1; l++) {
          if ((l == -1 && ibin == fW && bc.pW != PERIODIC) ||
              (l ==  1 && ibin == fE && bc.pE != PERIODIC) ||
              (l == -1 && ibin == fW && bc.pW == PERIODIC && xnBin == 0) ||
              (l ==  1 && ibin == fE && bc.pE == PERIODIC && xnBin == 0)) {
            continue;
          } else if (l == -1 && ibin == fW && bc.pW == PERIODIC) {
            iStride = fE;
          } else if (l ==  1 && ibin == fE && bc.pE == PERIODIC) {
            iStride = fW;
          } else {
            iStride = ibin + l;
          }

          adjBin = iStride + jStride + kStride; 
          adjStart = binStart[adjBin];        // find start and end of bins
          adjEnd = binEnd[adjBin];
          if (adjStart != -1) {               // if bin is not empty
            for (target = adjStart; target < adjEnd; target++) {
              j = partInd[target];
              if (j != i) {                   // if its not original part

                // Add particle j to neighborList
                int nStride = i*nMax + cc;    // increment a counter
                neighborList[nStride] = j;
                neighborCount[i]++;
                cc++;
              }
            }
          }
        }
      }
    }
  }
}

__global__ void choose3(int *neighborCount, int *nChoose3, int nparts)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < nparts) {
    int c = neighborCount[i];
    nChoose3[i] = c*(c-1)*(c-2)/6;
  }
}

__global__ void combine_nodes(int *neighborList, int *neighborCount,
  int *nodes, int *strides, int nparts, int nMax)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int N1, N2, N3, N4;
  int countPerm = 0;
  int quartStride;


  if (i < nparts) {
    N1 = i;
    for (int j = 0; j < neighborCount[i]; j++) {
      N2 = neighborList[i*nMax + j];
      for (int k = j + 1; k < neighborCount[i]; k++) {
        N3 = neighborList[i*nMax + k];
        for (int l = k + 1; l < neighborCount[i]; l++) {
          N4 = neighborList[i*nMax + l];

          quartStride = 4*strides[i] + 4*countPerm;
          countPerm++;

          nodes[quartStride] = N1;
          nodes[quartStride + 1] = N2;
          nodes[quartStride + 2] = N3;
          nodes[quartStride + 3] = N4;
        }
      }
    }
  }
}

__global__ void sort_combos(int *nodes, int nPerms)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  int startInd;
  int vals[4];

  if (i < nPerms) {
    startInd = 4*i; 
    for (int v = 0; v < 4; v++) {
      vals[v] = nodes[startInd + v];
    }
    // Adapated from stackoverflow.com 2786899, 
    //  "Fastest sort of fixed length 6 int array"
    // and "pages.ripco.net/~jgamble/nw.html"
    // and perhaps "Bit Twiddling Hacks" by Sean Eron Anderson @ Stanford
    #define min(x, y) (x*(x < y) + y*(y < x) + x*(x == y))
    #define max(x, y) (x*(x > y) + y*(y > x) + y*(x == y))
    #define SWAP(x,y) { const int a = min(vals[x], vals[y]); \
                        const int b = max(vals[x], vals[y]); \
                        vals[x] = a; vals[y] = b; }
    SWAP(0,1);
    SWAP(2,3);
    SWAP(0,2);
    SWAP(1,3);
    SWAP(1,2);

    #undef SWAP
    #undef min
    #undef max

    for (int v = 0; v < 4; v++) {
      nodes[startInd + v] = vals[v];
    }
  }
}

__global__ void find_unique(int *nodes, int base, int remainder, int *isUnique)
{
  int threadInd = threadIdx.x + blockIdx.x*blockDim.x;

  // only want to look at OTHER sets of nodes, so add (n+1)
  int target = threadInd + (base + 1);// Target set
  int targetNode = 4*target;          // First node of target set

  int baseNode = 4*base;              // First node of base set
  int nSame = 0;

  if (isUnique[base] == 0) {    // if base is not unique, skip
      return;
  }
  __syncthreads();

  if (threadInd < remainder) {
    nSame = (nodes[baseNode] == nodes[targetNode]) 
          + (nodes[baseNode + 1] == nodes[targetNode + 1])
          + (nodes[baseNode + 2] == nodes[targetNode + 2]) 
          + (nodes[baseNode + 3] == nodes[targetNode + 3]);

    // set isUnique to 1 if nSame != 4, else set to zero
    // if isUnique == 1
    //  if nSame != 4 --> isUnique = 1*1 = 1
    //  if nSame == 4 --> isUnique = 0*1 = 0
    // if isUnique == 0
    //  if nSame != 4 --> isUnique = 1*0 = 0
    //  if nSame == 4 --> isUnique = 0*0 = 0
    isUnique[target] = (nSame != 4)*isUnique[target];
  }
}

__global__ void find_unique2(int *nodes, int *isUnique, int nPerms) {
  int base = threadIdx.x + blockIdx.x*blockDim.x;

  if (base < nPerms) {
    int baseNode = 4*base;
    int target, targetNode;
    int nSame = 0;
    for (target = 0; target < nPerms; target++) {
      targetNode = 4*target;

      nSame = (nodes[baseNode] == nodes[targetNode]) 
            + (nodes[baseNode + 1] == nodes[targetNode + 1])
            + (nodes[baseNode + 2] == nodes[targetNode + 2]) 
            + (nodes[baseNode + 3] == nodes[targetNode + 3]);
      // if target == base, set nSame to 1
      nSame = (target == base) + nSame*(target != base);

      // set isUnique to 1 if nSame != 4, else set to zero
      // if isUnique == 1
      //  if nSame != 4 --> isUnique = 1*1 = 1
      //  if nSame == 4 --> isUnique = 0*1 = 0
      // if isUnique == 0
      //  if nSame != 4 --> isUnique = 1*0 = 0
      //  if nSame == 4 --> isUnique = 0*0 = 0
      isUnique[base] *= (nSame != 4);
       
    }
  }
}

__global__ void pull_unique(int *uniqueNodes, int *nodes, int *isUnique, 
  int nPerms, int *uniquePrefix, int nUnique)
{
  int TID = threadIdx.x + blockIdx.x*blockDim.x;
  int ind;

  if (TID < nPerms) {
    // becomes uniquePrefix[TID] - 1 iff is unique
    // becomes nUnique iff is !unique
    ind = (uniquePrefix[TID] - 1) * isUnique[TID] 
        + nUnique*(1 - isUnique[TID]);

    uniqueNodes[4*ind] = nodes[4*TID];
    uniqueNodes[4*ind + 1] = nodes[4*TID + 1];
    uniqueNodes[4*ind + 2] = nodes[4*TID + 2];
    uniqueNodes[4*ind + 3] = nodes[4*TID + 3];

  }
}

__global__ void fill_nodes(tetrad_struct *tetrads, int *uniqueNodes, 
  int nUnique)
{
  int i = threadIdx.x + blockIdx.x*blockDim.x;

  if (i < nUnique) {
    tetrads[i].N1 = uniqueNodes[4*i];    
    tetrads[i].N2 = uniqueNodes[4*i + 1];    
    tetrads[i].N3 = uniqueNodes[4*i + 2];    
    tetrads[i].N4 = uniqueNodes[4*i + 3];    
    tetrads[i].tolCheck = 0;
  }
}

__global__ void tetrad_geometry(part_struct *parts, tetrad_struct *tetrads,
  dom_struct *dom, int nUnique)
{
  int tet = threadIdx.x + blockIdx.x*blockDim.x;

  /* Tetrahedron Geometry Variables */
  double XCM = 0;       // Tetrad center of mass -- x
  double YCM = 0;       // Tetrad center of mass -- y
  double ZCM = 0;       // Tetrad center of mass -- x
  double r1[nDim];      // Node 1 relative coordinates
  double r2[nDim];      // Node 2 relative coordinates
  double r3[nDim];      // Node 3 relative coordinates
  double r4[nDim];      // Node 4 relative coordinates

  /* Shape Tensor Variables */
  double g[nDim2];      // Gyration tensor
  double avgLambda;     // Average eigenvalue of g = trace(g)/3 
  double g_hat[nDim2];  // Deviatoric part of g

  /* Velocity variables */

  double gInv[nDim2];   // Gyration tensor inverse
  int nrot;

  double UCM = 0;       // Tetrad center of vel -- u
  double VCM = 0;       // Tetrad center of vel -- v
  double WCM = 0;       // Tetrad center of vel -- w
  double u1[nDim];      // Node 1 relative vel
  double u2[nDim];      // Node 2 relative vel
  double u3[nDim];      // Node 3 relative vel
  double u4[nDim];      // Node 4 relative vel
  double W[nDim2];      // Velocity tensor

  double M[nDim2];      // Coarse-grained vel grad tensor
  double S[nDim2];      // Symmetric part of M
  double O[nDim2];      // Anti-Symmetric part of M

  if (tet < nUnique) {
    /*  POSITION  */
    // Fix periodicity issues
    periodic_flip(r1, r2, r3, r4, tetrads[tet], parts, dom->xl, dom->yl, 
      dom->zl);

    // Calculate tetrad center of mass
    // reference all of them to N1, if > dom.size, flip it
    XCM = 0.25*(r1[0] + r2[0] + r3[0] + r4[0]);
    YCM = 0.25*(r1[1] + r2[1] + r3[1] + r4[1]);
    ZCM = 0.25*(r1[2] + r2[2] + r3[2] + r4[2]);

    // Relate nodes to center of mass
    r1[0] -= XCM;
    r1[1] -= YCM;
    r1[2] -= ZCM;
            
    r2[0] -= XCM;
    r2[1] -= YCM;
    r2[2] -= ZCM;
            
    r3[0] -= XCM;
    r3[1] -= YCM;
    r3[2] -= ZCM;
            
    r4[0] -= XCM;
    r4[1] -= YCM;
    r4[2] -= ZCM;

    // Gyration tensor
    for (int i = 0; i < nDim; i++) {
      for (int j = 0; j < nDim; j++) {
        g[nDim*i + j] = 0.25*(r1[i]*r1[j] + r2[i]*r2[j] 
                            + r3[i]*r3[j] + r4[i]*r4[j]);
      }
    }
    tetrads[tet].R2 = matrixTrace3(g);

    // Calculate average eigenvalue of g
    avgLambda = tetrads[tet].R2/3.;

    // Calculate deviatoric part of g, g_hat
    for (int i = 0; i < nDim; i++) {
      for (int j = 0; j < nDim; j++) {
        int c = nDim*i + j;
        g_hat[c] = g[c] - avgLambda*(i == j);
      }
    }

    // Calculate variance of g's eigenvalues, var = trace(g_hat^2)
    tetrads[tet].var = matrixSquaredTrace3(g_hat);

    // Calculate shape of g using det(g_hat)
    tetrads[tet].det = matrixDet3(g_hat);

    // Calculate I1, I2, I3 and principal directions of shape tensor
    jacobiEig3(g, tetrads[tet].gEigVal, tetrads[tet].gEigVec, &nrot);

    /* Velocity */
    // Reinit g since it was overwritten in last step
    for (int i = 0; i < nDim; i++) {
      for (int j = 0; j < nDim; j++) {
        g[nDim*i + j] = (r1[i]*r1[j] + r2[i]*r2[j] 
                       + r3[i]*r3[j] + r4[i]*r4[j]);
      }
    }

    // find gInv
    matrixInverse3(g, gInv);

    // Calculate tetrad center of vel
    int N1 = tetrads[tet].N1;
    int N2 = tetrads[tet].N2;
    int N3 = tetrads[tet].N3;
    int N4 = tetrads[tet].N4;

    UCM = 0.25*(parts[N1].u + parts[N2].u + parts[N3].u + parts[N4].u);
    VCM = 0.25*(parts[N1].v + parts[N2].v + parts[N3].v + parts[N4].v);
    WCM = 0.25*(parts[N1].w + parts[N2].w + parts[N3].w + parts[N4].w);

    // Relate nodes to center of vel
    u1[0] = parts[N1].u - UCM;
    u1[1] = parts[N1].v - VCM;
    u1[2] = parts[N1].w - WCM;

    u2[0] = parts[N2].u - UCM;
    u2[1] = parts[N2].v - VCM;
    u2[2] = parts[N2].w - WCM;

    u3[0] = parts[N3].u - UCM;
    u3[1] = parts[N3].v - VCM;
    u3[2] = parts[N3].w - WCM;

    u4[0] = parts[N4].u - UCM;
    u4[1] = parts[N4].v - VCM;
    u4[2] = parts[N4].w - WCM;

    // Vel tensor
    for (int i = 0; i < 2; i++) {
      for (int j = 0; j < 2; j++) {
        W[3*i + j] = r1[i]*u1[j] + r2[i]*u2[j] + r3[i]*u3[j] + r4[i]*u4[j];
      }
    }

    // Velocity Gradient Tensor
    matrixMult3(gInv, W, M);

    // Decompose M
    for (int i = 0; i < 3; i++) {
      for (int j = 0; j < 3; j++) {
        S[3*i + j] = 0.5*(M[3*i + j] + M[3*j + i]);
        O[3*i + j] = 0.5*(M[3*i + j] - M[3*j + i]);
      }
    }
    // Find princpal directions and values of strain tensor
    jacobiEig3(S, tetrads[tet].sEigVal, tetrads[tet].sEigVec, &nrot);

    // pull vorticity vector:
    // See AP, Fluid Dynamics, pages 38 + 40 -- O is -(7.53) bc (8.9)
    // TODO: need to check
    tetrads[tet].vorticity[0] = 2*O[3];
    tetrads[tet].vorticity[1] = 2*O[2];
    tetrads[tet].vorticity[2] = 2*O[7];

  }
}

__global__ void check_tolerances(tetrad_struct *tetrads, double varCutLow,
  double varCutHigh, double shapeCutLow, double shapeCutHigh, int nUnique)
{
  int tet = threadIdx.x + blockIdx.x*blockDim.x;
  if (tet < nUnique) {
    double eigVar = 1.5*tetrads[tet].var/(tetrads[tet].R2*tetrads[tet].R2);
    double shape = 27.*tetrads[tet].det/(tetrads[tet].R2*tetrads[tet].R2*
                                          tetrads[tet].R2);
    tetrads[tet].tolCheck = (eigVar >= varCutLow &&
                             eigVar <= varCutHigh &&
                             shape >= shapeCutLow &&
                             shape <= shapeCutHigh);
  }
}

__global__ void matrixTests(void)
{
  // TEST MATRIX FUNCTIONS
    double A[9];
    double a_in[9];
    double B[9];
    double R[9];
    double invA[9];
    double d[3];
    double v[9];
    int nrot = 0;

  // INITIALIZE MATRICES
    A[0] = 3.; A[1] = 3.; A[2] = 5.; 
    A[3] = 3.; A[4] = 4.; A[5] = 6.;
    A[6] = 5.; A[7] = 6.; A[8] = -6.;

    B[0] = 2; B[1] = 6; B[2] = 1; 
    B[3] = 2; B[4] = 6; B[5] = 1;
    B[6] = 2; B[7] = 6; B[8] = 1;

    R[0] = 0.; R[1] = 0.; R[2] = 0.; 
    R[3] = 0.; R[4] = 0.; R[5] = 0.;
    R[7] = 0.; R[7] = 0.; R[8] = 0.;

    invA[0] = 0.; invA[3] = 0.; invA[6] = 0.; 
    invA[1] = 0.; invA[4] = 0.; invA[7] = 0.;
    invA[2] = 0.; invA[5] = 0.; invA[8] = 0.;

    d[0] = 0.; d[1] = 0.; d[2] = 0.; 

    v[0] = 0.; v[1] = 0.; v[2] = 0.; 
    v[1] = 0.; v[4] = 0.; v[5] = 0.;
    v[2] = 0.; v[7] = 0.; v[8] = 0.;
    

    // Copy input matrix
    for (int i = 0; i < 3; i++) {
      for (int j = 0; j < 3; j++) {
        a_in[3*i + j] = A[3*i + j];
      }
    }

    matrixInverse3(A, invA);
    matrixMult3(A, B, R);

    jacobiEig3(a_in, d, v, &nrot);

    printf("A = %lf, %lf, %lf\n\
    %lf, %lf, %lf,\n\
    %lf, %lf, %lf\n", A[0], A[1], A[2],
         A[3], A[4], A[5], A[6], A[7], A[8]);

    printf("invA = %lf, %lf, %lf\n\
       %lf, %lf, %lf,\n\
       %lf, %lf, %lf\n", invA[0], invA[1], invA[2],
         invA[3], invA[4], invA[5], invA[6], invA[7], invA[8]);

    printf("R = %lf, %lf, %lf\n\
    %lf, %lf, %lf,\n\
    %lf, %lf, %lf\n", R[0], R[1], R[2],
         R[3], R[4], R[5], R[6], R[7], R[8]);

    printf("d = %lf, %lf, %lf; nrot = %d\n", d[0], d[1], d[2], nrot);

    printf("v = %lf, %lf, %lf\n\
    %lf, %lf, %lf,\n\
    %lf, %lf, %lf\n", v[0], v[1], v[2],
                      v[3], v[4], v[5], 
                      v[6], v[7], v[8]);
}

__device__ void periodic_flip(double *r1, double *r2, double *r3, double *r4, 
  tetrad_struct tetrads, part_struct *parts, double xl, double yl, double zl)
{
    int N1 = tetrads.N1;
    int N2 = tetrads.N2;
    int N3 = tetrads.N3;
    int N4 = tetrads.N4;
    // branchless min
    #define flip(s1,s2,l,i) \
      {const double standard = s1[i] - s2[i]; \
       const double standard2 = standard*standard; \
       const double var1 = s1[i] - (s2[i] + l); \
       const double var2 = s1[i] - (s2[i] - l); \
       s2[i] += l*((var1*var1 < standard2) - (var2*var2 < standard2)); }

    // Set up position vectors to each particle from origin
    r1[0] = parts[N1].x;
    r1[1] = parts[N1].y;
    r1[2] = parts[N1].z;

    r2[0] = parts[N2].x;
    r2[1] = parts[N2].y;
    r2[2] = parts[N2].z;

    r3[0] = parts[N3].x;
    r3[1] = parts[N3].y;
    r3[2] = parts[N3].z;

    r4[0] = parts[N4].x;
    r4[1] = parts[N4].y;
    r4[2] = parts[N4].z;

    /* X direction */
    flip(r1, r2, xl, 0);
    flip(r1, r3, xl, 0);
    flip(r1, r4, xl, 0);

    /* Y direction */
    flip(r1, r2, yl, 1);
    flip(r1, r3, yl, 1);
    flip(r1, r4, yl, 1);

    /* Z direction */
    flip(r1, r2, zl, 2);
    flip(r1, r3, zl, 2);
    flip(r1, r4, zl, 2);

  #undef flip
}

__device__ double matrixDet3(double *A)
{
  double detA;
  detA = A[0]*(A[8]*A[4] - A[5]*A[7])
       - A[3]*(A[8]*A[1] - A[2]*A[7])
       + A[6]*(A[5]*A[1] - A[2]*A[4]);

  return detA;
}

__device__ double matrixTrace3(double *A)
{
  double trace = 0;
  for (int i = 0; i < 3; i++) {
    trace += A[nDim*i + i];
  }
  return trace;
}

__device__ double matrixSquaredTrace3(double *A)
{
  // computes trace(A^2) by first calculating A^2
  double A2[9] = {0,0,0,0,0,0,0,0,0};

  matrixMult3(A,A,A2);

  double trace = matrixTrace3(A2);

  return trace;
}

__device__ void matrixInverse3(double *A, double *invA)
{
  //     | A11 A12 A13 |   | A[0] A[1] A[2] |
  // A = | A21 A22 A23 | = | A[3] A[4] A[5] |
  //     | A31 A32 A33 |   | A[6] A[7] A[8] |
  double detA = matrixDet3(A);

  double invDETA = 1/detA;             

  // If it's bigger than this, the matrix is probably singular
  // 1 if okay, 0 if not
  int detCheck = (invDETA < 1e10);

  invA[0] = A[8]*A[4] - A[5]*A[7];
  invA[1] = -(A[8]*A[3] - A[6]*A[5]);
  invA[2] = A[7]*A[3] - A[6]*A[4];
  invA[3] = -(A[8]*A[1] - A[7]*A[2]);
  invA[4] = A[8]*A[0] - A[2]*A[6];
  invA[5] = -(A[7]*A[0] - A[1]*A[6]);
  invA[6] = A[5]*A[1] - A[4]*A[2];
  invA[7] = -(A[5]*A[0] - A[3]*A[2]);
  invA[8] = A[4]*A[0] - A[1]*A[3];

  for (int i = 0; i < 9; i++) {
    invA[i] *= detCheck*invDETA;
  }
}

__device__ void matrixMult3(double *A, double *B, double *R)
{
  // R = A*B
  // Rij = Aim*Bmj
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      R[3*i + j] = 0;
      for (int m = 0; m < 3; m++) {
        R[3*i + j] += A[3*i + m]*B[3*m + j];
      }
    }
  }
}

__device__ void jacobiEig3(double *a, double *d, double *v, int *nrot)
{
  // Modified from Numerical Recipes
  int i,j,ip,iq;
  double thresh, theta, tau, t, sm, s, h, g, c;
  int n = 3;
  double b[3], z[3];

  // Initialize v to identity
  for (ip = 0; ip < n; ip++) {
    for (iq = 0; iq < n; iq++) {
      v[3*ip + iq] = (ip == iq);
    }
  }

  // Intialize b,d to diagonal of a; z to 0
  for (ip = 0; ip < n; ip++) {
    b[ip] = a[3*ip + ip];
    d[ip] = a[3*ip + ip];
    z[ip] = 0.;
  }

  // Main loop
  for (i = 1; i < 50; i++) {
    sm = 0.0;
    // Sum magnitude of off-diagonal elements of a
    for (ip = 0; ip < n-1; ip++) {
      for (iq = ip + 1; iq < n; iq++) {
        sm += abs(a[3*ip + iq]);
      }
    }

    // Normal return, relies on quadratic convergence to machine underflow
    if (sm == 0.0) {
      eigsrt(d,v);
      return;
    }

    if (i < 4) {
      thresh = 0.2*sm/(n*n);    // On first 3 sweeps...
    } else {
      thresh = 0.;               // ...thereafter
    }

    for (ip = 0; ip < n - 1; ip++) {
      for (iq = ip + 1; iq < n; iq++) {
        g = 100.0*abs(a[3*ip + iq]);

        // After 4 sweeps, skip the rotation if the off-diagonal element is sm.
        if (i > 4 && g <= DBL_EPSILON*abs(d[ip]) && g <= DBL_EPSILON*abs(d[iq])) {
          a[3*ip + iq] = 0; 
        } else if (abs(a[3*ip + iq]) > thresh) {
          h = d[iq] - d[ip];
          if (g <= DBL_EPSILON*abs(h)) {
            t = a[3*ip + iq]/h;   // t = 1/(2theta)
          } else {
            theta = 0.5*h/a[3*ip + iq];
            t = 1.0/(abs(theta) + sqrt(1.0 + theta*theta));
            if (theta < 0.0) {
              t = -t;
            }
          }
          c = 1.0/sqrt(1.0 + t*t);
          s = t*c;
          tau = s/(1.0 + c);
          h = t*a[3*ip + iq];
          z[ip] -= h;
          z[iq] += h;
          d[ip] -= h;
          d[iq] += h;
          a[3*ip + iq] = 0.0;
          for (j = 0; j < ip; j++) {  // Case 0 <= j < p
            rot(a,s,tau,j,ip,j,iq);
          }
          for (j = ip + 1; j < iq; j++) {  // Case p < j < q
            rot(a,s,tau,ip,j,j,iq);
          }
          for (j = iq + 1; j < n; j++) {  // Case q < j < n
            rot(a,s,tau,ip,j,iq,j);
          }
          for (j = 0; j < n; j++) {  // Case 0 <= j < p
            rot(v,s,tau,j,ip,j,iq);
          }
          (*nrot)++;
        }
      }
    }

    // Update d with sum of t*a_pq and reinit z
    for (ip = 0; ip < n; ip++) {
      b[ip] += z[ip];
      d[ip] = b[ip];
      z[ip] = 0.0;
    }
  }
//  //printf("Too many iterations in routine jacobi\n");
//  //return EXIT_FAILURE;
}

// rotation kernel
__device__ void rot(double *a, double s, double tau, int i, int j, int k, 
  int l)
{
  double g = a[3*i + j];
  double h = a[3*k + l];
  a[3*i + j] = g - s*(h + g*tau);
  a[3*k + l] = h + s*(g - h*tau);
}

// sort eigen values and eigenvectors into descending order
__device__ void eigsrt(double *d, double *v)
{
  int k;
  int n = 3;
  double p;

  // Loop over all eigenvalues (find max)
  for (int i = 0; i < n - 1; i++) {
    k = i;      // Index of currEig
    p = d[k];   // Value of currEig

    // Loop over all other eigenvalues
    for (int j = i; j < n; j++) {

      // If targetEig > currEig
      if (d[j] >= p) {
        k = j;      // Index of targetEig
        p = d[k];   // Value of targetEig
      }
    }

    // If we've found a targetEig > currEig
    if (k != i) {
      d[k] = d[i];  // Set value of targetEig to currEig
      d[i] = p;     // Set value of currEig to targetEig

      // Arrange eigenvectors
      for (int j = 0; j < n; j++) {
        p = v[3*j + i];
        v[3*j + i] = v[3*j + k];
        v[3*j + k] = p;
      }
    }
  }
}
