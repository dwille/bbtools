#include "hip/hip_runtime.h"
#include "cuda_sort.h"
#include "time.h"

#include <hip/hip_runtime.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/sort.h>
#include <thrust/scan.h>


extern "C"
void cuda_dev_pull(void)
{
//  hipMemcpy(neighborList, _neighborList, nMax*nparts*sizeof(int),
//    hipMemcpyDeviceToHost);
}

extern "C"
void cuda_dom_push(void)
{
  hipSetDevice(dev_start);
  // copy host data to device
  hipMemcpy(_dom, &dom, sizeof(dom_struct), hipMemcpyHostToDevice);
  hipMemcpy(_binDom, &binDom, sizeof(dom_struct), hipMemcpyHostToDevice);
}

extern "C"
void cuda_part_push(void)
{
  hipSetDevice(dev_start);
  hipMemcpy(_parts, parts, sizeof(part_struct) * nparts, 
    hipMemcpyHostToDevice);
}

extern "C"
void cuda_dev_malloc(void)
{
  // allocate device memory on device
  hipSetDevice(dev_start);

  hipMalloc((void**) &(_parts), sizeof(part_struct) * nparts);
  hipMalloc((void**) &(_dom), sizeof(dom_struct));
  hipMalloc((void**) &(_binDom), sizeof(dom_struct));
}

void cuda_find_tetrads()
{
  // set up cuda threads and blocks
  int threads = MAX_THREADS_1D;
  int blocks = (int) ceil((double) nparts / (double) threads);
  if (threads > nparts) {
    threads = nparts;
    blocks = 1;
  }
  dim3 dimBlocks(threads);
  dim3 numBlocks(blocks);

  // set up bins and search for tetrads
  if (nparts < 4) {
    printf("nparts = %d, no tetrads to find.\n", nparts);
    exit(EXIT_FAILURE);
  } else if (nparts >= 4) {

    int nBins = binDom.Gcc.s3;

    // initialize threads for nBin size
    int threads_nb = MAX_THREADS_1D;
    int blocks_nb = (int) ceil((double) nBins / (double) threads_nb);
    if (threads_nb > nBins) {
      threads_nb = nBins;
      blocks_nb = 1;
    }
    dim3 dimBlocks_nb(threads_nb);
    dim3 numBlocks_nb(blocks_nb);

    // Go to each particle and find its bin
    int *_partInd;
    int *_partBin;

    hipMalloc((void**) &_partInd, nparts*sizeof(int));
    hipMalloc((void**) &_partBin, nparts*sizeof(int));
    bin_fill<<<numBlocks, dimBlocks>>>(_partInd, _partBin, nparts,
      _parts, _binDom, bc);
      
    /* sort by bin */
    thrust::device_ptr<int> ptr_partBin(_partBin);
    thrust::device_ptr<int> ptr_partInd(_partInd);
    thrust::sort_by_key(ptr_partBin, ptr_partBin + nparts, ptr_partInd);
    _partBin = thrust::raw_pointer_cast(ptr_partBin);
    _partInd = thrust::raw_pointer_cast(ptr_partInd);

    /* calculate start and end index of each bin */
    int *_binStart;
    int *_binEnd;
    hipMalloc((void**) &_binStart, nBins*sizeof(int));
    hipMalloc((void**) &_binEnd, nBins*sizeof(int));
    init<<<numBlocks_nb, dimBlocks_nb>>>(_binStart, nBins, -1);
    init<<<numBlocks_nb, dimBlocks_nb>>>(_binEnd, nBins, -1);

    int smemSize = sizeof(int)*(threads + 1);
    bin_start<<<blocks, threads, smemSize>>>(_binStart, _binEnd, _partBin,
      nparts);

    /*  FIND_NODES */
    // Find all neighbors in adjacent bins for each particle; count them
    int *_neighborList;
    int *_neighborCount;
    hipMalloc((void**) &_neighborList, nMax*nparts*sizeof(int));
    hipMalloc((void**) &_neighborCount, nparts*sizeof(int));
    init<<<numBlocks, dimBlocks>>>(_neighborCount, nparts, 0);

    printf("\tFinding possible tetrad permutations... ");
    find_nodes<<<numBlocks, dimBlocks>>>(_parts, nparts, _dom, bc, _binStart,
      _binEnd, _partBin, _partInd, _binDom, _neighborList, _neighborCount, 
      nMax);

    /*  CHOOSE3 */
    int *_nChoose3;
    hipMalloc((void **) &_nChoose3, nparts*sizeof(int));
    choose3<<<numBlocks, dimBlocks>>>(_neighborCount, _nChoose3, nparts);

    // Find total number of permutations -- sum _nChoose3
    thrust::device_ptr<int> ptr_nChoose3(_nChoose3);
    int nPerms = thrust::reduce(ptr_nChoose3, 
      ptr_nChoose3 + nparts);
    int totalNodes = 4*nPerms;
    printf("Found %d.\n", nPerms);

    // Find stride for each particle
    int *_strides;
    hipMalloc((void **) &_strides, nparts*sizeof(int));
    thrust::device_ptr<int> ptr_strides(_strides);
    thrust::exclusive_scan(ptr_nChoose3, ptr_nChoose3 + nparts, ptr_strides);

    // create array to hold particle index of tetrad nodes
    int *_nodes;
    hipMalloc((void **) &_nodes, totalNodes*sizeof(int));

    int threads_nodes = MAX_THREADS_1D;
    int blocks_nodes = (int) ceil((double) totalNodes / (double) threads_nodes);
    if (threads_nodes > totalNodes) {
      threads_nodes = totalNodes;
      blocks_nodes = 1;
    }
    dim3 dimBlocks_nodes(threads_nodes);
    dim3 numBlocks_nodes(blocks_nodes);

    init<<<numBlocks_nodes, dimBlocks_nodes>>>(_nodes, totalNodes, -1);

    /*  COMBINE_NODES */
    // parallelizing over all particles, find all combitions for each particle
    printf("\tCombining nodes... ");
    combine_nodes<<<numBlocks, dimBlocks>>>(_neighborList, _neighborCount, 
      _nodes, _strides, nparts, nMax);

    /*  SORT_COMBOS */
    // Parallelizing over all permutations, sort each
    int threads_perms = MAX_THREADS_1D;
    int blocks_perms = (int) ceil((double) nPerms / (double) threads_perms);
    if (threads_perms > nPerms) {
      threads_perms = nPerms;
      blocks_perms = 1;
    }
    dim3 dimBlocks_perms(threads_perms);
    dim3 numBlocks_perms(blocks_perms);

    printf("Done!\n\tSorting permutations... ");
    sort_combos<<<numBlocks_perms, dimBlocks_perms>>>(_nodes, nPerms);

    /*  FIND_UNIQUE */
    // compare and find unique ones
    int *_isUnique;
    hipMalloc((void **) &_isUnique, nPerms*sizeof(int));
    init<<<numBlocks_perms, dimBlocks_perms>>>(_isUnique, nPerms, 1);

    // Loop over each permutations, then parallelize over the remaining
    printf("Done!\n\tLooping over permutations and finding unique sets...");
    for (int base = 0; base < (nPerms - 1); base++) {

      // set up threads and blocks
      int remainder = nPerms - base - 1;
      int remT = MAX_THREADS_1D;
      int remB = (int) ceil((double) remainder / (double) remT);
      if (remT > remainder) {
        remT = remainder;
        remB = 1;
      }
      dim3 dimBlocks_rem(remT);
      dim3 numBlocks_rem(remB);

      // determine whether target node is a duplicate, mark if so
      find_unique<<<numBlocks_rem, dimBlocks_rem>>>(_nodes, base, remainder, 
        _isUnique);  
    }
//    find_unique2<<<numBlocks_perms, dimBlocks_perms>>>(_nodes, _isUnique, 
//      nPerms);

    // sum to find number of unique combinations
    thrust::device_ptr<int> ptr_isUnique(_isUnique);
    int nUnique = thrust::reduce(ptr_isUnique, ptr_isUnique + nPerms);

    printf("Found %d\n", nUnique);

    /*  PULL UNIQUE NODES */
    // Last entry is trash for finding indices and redirecting
    int *_uniqueNodes;
    hipMalloc((void**) &_uniqueNodes, 4*(nUnique + 1)*sizeof(int));

    int threadsU = MAX_THREADS_1D;
    int blocksU = (int) ceil((double) 4*(nUnique + 1) / (double) threadsU);
    if (threadsU > 4*(nUnique + 1)) {
      threadsU = 4*(nUnique + 1);
      blocksU = 1;
    }
    dim3 dimBlocks_U(threadsU);
    dim3 numBlocks_U(blocksU);
    init<<<numBlocks_U, dimBlocks_U>>>(_uniqueNodes, 4*(nUnique + 1), -1);

    // Prefix sum of _isUnique
    int *_uniquePrefix;
    hipMalloc((void **) &_uniquePrefix, nPerms*sizeof(int));
    thrust::device_ptr<int> ptr_uPref(_uniquePrefix);
    thrust::device_ptr<int> ptr_isUn(_isUnique);
    thrust::inclusive_scan(ptr_isUn, ptr_isUn + nPerms, ptr_uPref);

    printf("\tPulling unique nodes... ");
    pull_unique<<<numBlocks_perms, dimBlocks_perms>>>(_uniqueNodes, _nodes, 
      _isUnique, nPerms, _uniquePrefix, nUnique);
    printf("Done!\n");
  
    /* FIND REGULAR */
    // Initialize tetrad struct for all unique tetrads
    tetrad_struct *_allTetrads;
    hipMalloc((void**) &(_allTetrads), sizeof(tetrad_struct) * nUnique);

    // Set up threads, blocks for each tetrad
    int threads_tetrads = MAX_THREADS_1D;
    int blocks_tetrads = (int) ceil((double) nUnique /(double) threads_tetrads);
    if (threads_tetrads > nUnique) {
      threads_tetrads = nUnique;
      blocks_tetrads = 1;
    }
    dim3 dimBlocks_tetrads(threads_tetrads);
    dim3 numBlocks_tetrads(blocks_tetrads);

    // Init isRegular array
    printf("\tFinding regular tetrads... ");
    int *_isRegular;
    hipMalloc((void**) &(_isRegular), nUnique * sizeof(int));

    // Fill _allTetrads with the correct nodes and init isRegular
    fill_nodes<<<numBlocks_tetrads, dimBlocks_tetrads>>>(_allTetrads,
      _uniqueNodes, _isRegular, nUnique);

    // Tolerance check on all tetrads
    check_tolerances<<<numBlocks_tetrads, dimBlocks_tetrads>>>(_parts, 
      _allTetrads, _dom, _isRegular, nUnique, varCutLow, varCutHigh,
      shapeCutLow, shapeCutHigh);

    // Find number of tetrads that meet the regularity tolerance
    thrust::device_ptr<int> ptr_isReg(_isRegular);
    nRegular = thrust::reduce(ptr_isReg, ptr_isReg + nUnique);
    printf("Found %d\n", nRegular);

    printf("\tIntializing regular tetrads... ");
    
    // Prefix sum on _isRegular -- will give indices for smaller array
    int *_regularPrefix;
    hipMalloc((void **) &(_regularPrefix), nUnique * sizeof(int));
    thrust::device_ptr<int> ptr_rPref(_regularPrefix);
    thrust::inclusive_scan(ptr_isReg, ptr_isReg + nUnique, ptr_rPref);

    // Initialize array to hold indices of regular tetrads
    // -- last index is trash for redirecting output
    int *_regularTetrads;
    hipMalloc((void**) &(_regularTetrads), (nRegular + 1) * sizeof(int));

    // Pull regular tetrads
    pull_regular<<<numBlocks_tetrads, dimBlocks_tetrads>>>(_regularTetrads,
      _isRegular, _regularPrefix, nUnique, nRegular);

    // Set up threads, blocks for each regular tetrad
    int threads_regular = MAX_THREADS_1D;
    int blocks_regular = (int) ceil((double) nRegular/(double) threads_regular);
    if (threads_regular > nRegular) {
      threads_regular = nRegular;
      blocks_regular = 1;
    }
    dim3 dimBlocks_regular(threads_regular);
    dim3 numBlocks_regular(blocks_regular);

    // Alloc new tetrad struct, and pull indices / nodes
    hipMalloc((void**) &_tetrads, sizeof(tetrad_struct) * nRegular);
    copy_regular<<<numBlocks_regular, dimBlocks_regular>>>(_tetrads, 
      _allTetrads, _regularTetrads, nRegular, _isRegular);

    printf("Done.\n");

    // Free variables
    hipFree(_partInd);
    hipFree(_partBin);
    hipFree(_binStart);
    hipFree(_binEnd);
    hipFree(_neighborCount);
    hipFree(_neighborList);
    hipFree(_nChoose3);
    hipFree(_strides);
    hipFree(_nodes);
    hipFree(_uniquePrefix);
    hipFree(_isUnique);
    hipFree(_uniqueNodes);

    hipFree(_isRegular);
    hipFree(_allTetrads);
    hipFree(_regularPrefix);
    hipFree(_regularTetrads);
  }
}      

extern "C"
void cuda_tetrad_malloc(void)
{
  // Allocate tetrad struct on host and pull from device
  tetrads = (tetrad_struct*) malloc(nRegular * sizeof(tetrad_struct));
  // Pull tetrads back to host
  hipMemcpy(tetrads, _tetrads, nRegular * sizeof(tetrad_struct), 
      hipMemcpyDeviceToHost);

  hipSetDevice(dev_start);

  hipMalloc((void**) &(_R2), sizeof(double) * nRegular);
  hipMalloc((void**) &(_var), sizeof(double) * nRegular);
  hipMalloc((void**) &(_shape), sizeof(double) * nRegular);

  hipMalloc((void**) &(_gEigVal), 3 * sizeof(double) * nRegular);
  hipMalloc((void**) &(_gEigVec), 9 * sizeof(double) * nRegular);
  hipMalloc((void**) &(_sEigVal), 3 * sizeof(double) * nRegular);
  hipMalloc((void**) &(_sEigVec), 9 * sizeof(double) * nRegular);
  hipMalloc((void**) &(_vorticity), 3 * sizeof(double) * nRegular);
  hipMalloc((void**) &(_vortMag), sizeof(double) * nRegular);
  
  hipMalloc((void**) &(_gEigVecInit), 9 * sizeof(double) * nRegular);
  hipMalloc((void**) &(_sEigVecInit), 9 * sizeof(double) * nRegular);


  hipMalloc((void**) &(_g1_s1), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g1_s2), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g1_s3), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g2_s1), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g2_s2), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g2_s3), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g3_s1), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g3_s2), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g3_s3), sizeof(double) * nRegular);

  hipMalloc((void**) &(_g1_z), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g2_z), sizeof(double) * nRegular);
  hipMalloc((void**) &(_g3_z), sizeof(double) * nRegular);
  hipMalloc((void**) &(_s1_z), sizeof(double) * nRegular);
  hipMalloc((void**) &(_s2_z), sizeof(double) * nRegular);
  hipMalloc((void**) &(_s3_z), sizeof(double) * nRegular);
  hipMalloc((void**) &(_w_z), sizeof(double) * nRegular);

  hipMalloc((void**) &(_w_g1), sizeof(double) * nRegular);
  hipMalloc((void**) &(_w_g2), sizeof(double) * nRegular);
  hipMalloc((void**) &(_w_g3), sizeof(double) * nRegular);
  hipMalloc((void**) &(_w_s1), sizeof(double) * nRegular);
  hipMalloc((void**) &(_w_s2), sizeof(double) * nRegular);
  hipMalloc((void**) &(_w_s3), sizeof(double) * nRegular);
}

void cuda_tetrad_stats(void)
{ 
  // Matrix tests
  #ifdef DEBUG
    if (tt == 0) {
      matrixTests<<<1,1>>>();
    }
  #endif
  // Parallelize over each tetrad
  int threads_tetrads = MAX_THREADS_1D;
  int blocks_tetrads = (int) ceil((double) nRegular / (double) threads_tetrads);
  if (threads_tetrads > nRegular) {
    threads_tetrads = nRegular;
    blocks_tetrads = 1;
  }
  dim3 dimBlocks_tetrads(threads_tetrads);
  dim3 numBlocks_tetrads(blocks_tetrads);

  // Calculate tetrad geometry and velocity measures
  tetrad_geometry<<<numBlocks_tetrads, dimBlocks_tetrads>>>(_parts, _tetrads,
    _dom, _R2, _var, _shape, _gEigVal, _gEigVec, _sEigVal, _sEigVec, 
    _vorticity, _vortMag, nRegular);

  // If first timestep, save vectors for later comparison
  if (tt == 0) {
    hipMemcpy(_gEigVecInit, _gEigVec, 9*sizeof(double)*nRegular,
      hipMemcpyDeviceToDevice);
    hipMemcpy(_sEigVecInit, _sEigVec, 9*sizeof(double)*nRegular,
      hipMemcpyDeviceToDevice);
  }

  // Copy back to host for writing to file
  hipMemcpy(R2, _R2, sizeof(double) * nRegular, hipMemcpyDeviceToHost);
  hipMemcpy(var, _var, sizeof(double) * nRegular, hipMemcpyDeviceToHost);
  hipMemcpy(shape, _shape, sizeof(double) * nRegular, hipMemcpyDeviceToHost);

  hipMemcpy(gEigVal, _gEigVal, 3 * sizeof(double) * nRegular, 
    hipMemcpyDeviceToHost);
  hipMemcpy(gEigVec, _gEigVec, 9 * sizeof(double) * nRegular, 
    hipMemcpyDeviceToHost);
  hipMemcpy(sEigVal, _sEigVal, 3 * sizeof(double) * nRegular, 
    hipMemcpyDeviceToHost);
  hipMemcpy(sEigVec, _sEigVec, 9 * sizeof(double) * nRegular, 
    hipMemcpyDeviceToHost);
  hipMemcpy(vorticity, _vorticity, 3 * sizeof(double) * nRegular, 
    hipMemcpyDeviceToHost);

  // Calculate means of scalar parameters
  double inRegular = 1./nRegular;

  thrust::device_ptr<double> ptr_R2(_R2);
  thrust::device_ptr<double> ptr_var(_var);
  thrust::device_ptr<double> ptr_shape(_shape);
  thrust::device_ptr<double> ptr_vortMag(_vortMag);

  meanR2 = thrust::reduce(ptr_R2, ptr_R2 + nRegular) * inRegular;
  meanVar = thrust::reduce(ptr_var, ptr_var + nRegular) * inRegular;
  meanShape = thrust::reduce(ptr_shape, ptr_shape + nRegular) * inRegular;
  mean_vortMag= thrust::reduce(ptr_vortMag, ptr_vortMag+nRegular)*inRegular; 

  // Calculate std of scalar parameters
  scalar_std<<<numBlocks_tetrads, dimBlocks_tetrads>>>(_R2, _var, _shape,
    meanR2, meanVar, meanShape, nRegular);

  stdR2 = sqrt(thrust::reduce(ptr_R2, ptr_R2 + nRegular));
  stdVar = sqrt(thrust::reduce(ptr_var, ptr_var + nRegular));
  stdShape = sqrt(thrust::reduce(ptr_shape, ptr_shape + nRegular));

  // Calculate alignment of vectors
  align_vectors<<<numBlocks_tetrads, dimBlocks_tetrads>>>(_gEigVec, _sEigVec,
    _vorticity, _gEigVecInit, _sEigVecInit, nRegular,
    _g1_s1, _g1_s2, _g1_s3,
    _g2_s1, _g2_s2, _g2_s3,
    _g3_s1, _g3_s2, _g3_s3,
    _g1_z, _g2_z, _g3_z,
    _s1_z, _s2_z, _s3_z,
    _w_z,
    _w_g1, _w_g2, _w_g3,
    _w_s1, _w_s2, _w_s3);

  // Calculate alignment means
  thrust::device_ptr<double> ptr_g1_s1(_g1_s1);
  thrust::device_ptr<double> ptr_g1_s2(_g1_s2);
  thrust::device_ptr<double> ptr_g1_s3(_g1_s3);
  thrust::device_ptr<double> ptr_g2_s1(_g2_s1);
  thrust::device_ptr<double> ptr_g2_s2(_g2_s2);
  thrust::device_ptr<double> ptr_g2_s3(_g2_s3);
  thrust::device_ptr<double> ptr_g3_s1(_g3_s1);
  thrust::device_ptr<double> ptr_g3_s2(_g3_s2);
  thrust::device_ptr<double> ptr_g3_s3(_g3_s3);

  thrust::device_ptr<double> ptr_g1_z(_g1_z);
  thrust::device_ptr<double> ptr_g2_z(_g2_z);
  thrust::device_ptr<double> ptr_g3_z(_g3_z);
  thrust::device_ptr<double> ptr_s1_z(_s1_z);
  thrust::device_ptr<double> ptr_s2_z(_s2_z);
  thrust::device_ptr<double> ptr_s3_z(_s3_z);
  thrust::device_ptr<double> ptr_w_z(_w_z);

  thrust::device_ptr<double> ptr_w_g1(_w_g1);
  thrust::device_ptr<double> ptr_w_g2(_w_g2);
  thrust::device_ptr<double> ptr_w_g3(_w_g3);
  thrust::device_ptr<double> ptr_w_s1(_w_s1);
  thrust::device_ptr<double> ptr_w_s2(_w_s2);
  thrust::device_ptr<double> ptr_w_s3(_w_s3);

  mean_g1_s1 = thrust::reduce(ptr_g1_s1, ptr_g1_s1 + nRegular) * inRegular; 
  mean_g1_s2 = thrust::reduce(ptr_g1_s2, ptr_g1_s2 + nRegular) * inRegular; 
  mean_g1_s3 = thrust::reduce(ptr_g1_s3, ptr_g1_s3 + nRegular) * inRegular; 
  mean_g2_s1 = thrust::reduce(ptr_g2_s1, ptr_g2_s1 + nRegular) * inRegular; 
  mean_g2_s2 = thrust::reduce(ptr_g2_s2, ptr_g2_s2 + nRegular) * inRegular; 
  mean_g2_s3 = thrust::reduce(ptr_g2_s3, ptr_g2_s3 + nRegular) * inRegular; 
  mean_g3_s1 = thrust::reduce(ptr_g3_s1, ptr_g3_s1 + nRegular) * inRegular; 
  mean_g3_s2 = thrust::reduce(ptr_g3_s2, ptr_g3_s2 + nRegular) * inRegular; 
  mean_g3_s3 = thrust::reduce(ptr_g3_s3, ptr_g3_s3 + nRegular) * inRegular; 

  mean_g1_z  = thrust::reduce(ptr_g1_z, ptr_g1_z + nRegular) * inRegular; 
  mean_g2_z  = thrust::reduce(ptr_g2_z, ptr_g2_z + nRegular) * inRegular; 
  mean_g3_z  = thrust::reduce(ptr_g3_z, ptr_g3_z + nRegular) * inRegular; 
  mean_s1_z  = thrust::reduce(ptr_s1_z, ptr_s1_z + nRegular) * inRegular; 
  mean_s2_z  = thrust::reduce(ptr_s2_z, ptr_s2_z + nRegular) * inRegular; 
  mean_s3_z  = thrust::reduce(ptr_s3_z, ptr_s3_z + nRegular) * inRegular; 
  mean_w_z   = thrust::reduce(ptr_w_z, ptr_w_z + nRegular) * inRegular; 

  mean_w_g1  = thrust::reduce(ptr_w_g1, ptr_w_g1 + nRegular) * inRegular; 
  mean_w_g2  = thrust::reduce(ptr_w_g2, ptr_w_g2 + nRegular) * inRegular; 
  mean_w_g3  = thrust::reduce(ptr_w_g3, ptr_w_g3 + nRegular) * inRegular; 
  mean_w_s1  = thrust::reduce(ptr_w_s1, ptr_w_s1 + nRegular) * inRegular; 
  mean_w_s2  = thrust::reduce(ptr_w_s2, ptr_w_s2 + nRegular) * inRegular; 
  mean_w_s3  = thrust::reduce(ptr_w_s3, ptr_w_s3 + nRegular) * inRegular; 

  // Copy back alignment sructure -- will be necessary for hist, but not now
//  hipMemcpy(tetAlign, _tetAlign, sizeof(align_struct) * nRegular,
//    hipMemcpyDeviceToHost);
}

extern "C"
void cuda_dev_free(void)
{
  hipFree(_parts);
  hipFree(_dom);
  hipFree(_binDom);
  hipFree(_tetrads);

  hipFree(_R2);
  hipFree(_var);
  hipFree(_shape);
  hipFree(_gEigVal);
  hipFree(_gEigVec);
  hipFree(_sEigVal);
  hipFree(_sEigVec);
  hipFree(_vorticity);
  hipFree(_vortMag);

  hipFree(_gEigVecInit);
  hipFree(_sEigVecInit);


  hipFree(_g1_s1);
  hipFree(_g1_s2);
  hipFree(_g1_s3);
  hipFree(_g2_s1);
  hipFree(_g2_s2);
  hipFree(_g2_s3);
  hipFree(_g3_s1);
  hipFree(_g3_s2);
  hipFree(_g3_s3);

  hipFree(_g1_z);
  hipFree(_g2_z);
  hipFree(_g3_z);
  hipFree(_s1_z);
  hipFree(_s2_z);
  hipFree(_s3_z);
  hipFree(_w_z);

  hipFree(_w_g1);
  hipFree(_w_g2);
  hipFree(_w_g3);
  hipFree(_w_s1);
  hipFree(_w_s2);
  hipFree(_w_s3);

  hipDeviceReset();
}
