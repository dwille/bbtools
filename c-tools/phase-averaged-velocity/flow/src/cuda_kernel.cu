#include "hip/hip_runtime.h"
#include "myCuda.h"

// create phase mask, multiply by vel
__global__ void phase_mask(double *uf, double *vf, double *wf, int *phase, 
  int N3)
{
  int pp = threadIdx.x + blockIdx.x*blockDim.x;;

  if (pp < N3) {
    phase[pp] = (phase[pp] == -1);
    uf[pp] *= phase[pp];
    vf[pp] *= phase[pp];
    wf[pp] *= phase[pp];
  }
}
